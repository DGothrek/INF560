#include "hip/hip_runtime.h"
/**
 * File containing the 3 apply-filter functions implemented 
 * on the GPU using CUDA.
 */

#include <string.h>
#include <stdio.h>
#include <stdlib.h>

#include <datatype.h>

/********************************************************
 * Device functions *************************************
 ********************************************************/
__global__ void gray(im, height, width) {
    int i, j, moy, pos;
    i = threadIdx.x + blockIdx.x*blockDim.x;
    j = threadIdx.y + blockIdx.y*blockDim.y;
    // int id = threadIdx.x
    //         + blockDim.x*threadIdx.y
    //         + blockIdx.x*blockDim.x*blockDim.y
    //         + blockIdx.y*blockDim.x*blockDim.y*gridDim.x;

    pos = i + j*width;
    if (i < height && j < width) {
        moy = (im[pos].r + im[pos].g + im[pos].b) / 3;
        if (moy < 0) moy = 0;
        if (moy > 255) moy = 255;

        im[pos].r = moy;
        im[pos].g = moy;
        im[pos].b = moy;

    }
}

__global__ void blur() {

}

__global__ void sobel(im, im_new, height, width) {
    // a écrire
    
}

/********************************************************
 * Host functions****************************************
 ********************************************************/

void apply_gray_filter_gpu(animated_gif *image) {
    /**
     * Assuming images of same size in a multiple image GIF
     **/ 
    int print_time = 1;
    struct timeval t1, t2;

    int im_num;
    int width = image->width[0];
    int height = image->height[0];
    int size = width*height;

    /**
     * Allocation on the device once for all the images (if multiple)
     * Memory allocation + dimension of grid
     **/
    if (print_time) gettimeofday(&t1, NULL);

    pixel *device_image;
    hipMalloc(&device_image, size*sizeof(pixel));

    // Image cut into n*m rectangles
    int n = width / 1024 + 1;
    int m = height / 1024 + 1;
    dim3 dimGrid(n, m);
    dim3 dimBlock(width / n + 1, height / m + 1);

    if (print_time) {
        gettimeofday(&t2, NULL);
        printf("Alloc done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
        gettimeofday(&t1, NULL); 
    }

    /**
     * Memory transfert + computation
     **/
    for (im_num = 0; im_num < image->n_images; im_num++) {
        // Memory transfer
        hipMemcpy(device_image, image->p[im_num], size * sizeof(pixel), hipMemcpyHostToDevice);
        if (print_time) {
            gettimeofday(&t2, NULL);
            printf("Transfer done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
            gettimeofday(&t1, NULL); 
        }

        // Computation
        gray<<<dimGrid, dimBlock>>>(device_image, height, width);
        if (print_time) {
            gettimeofday(&t2, NULL);
            printf("Computation done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
            gettimeofday(&t1, NULL); 
        }

        // Transfer back
        hipMemcpy(image->p[im_num], device_image, size * sizeof(pixel), hipMemcpyHostToDevice);
        if (print_time) {
            gettimeofday(&t2, NULL);
            printf("Transfer back done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
            gettimeofday(&t1, NULL); 
        }

    }

    hipFree(device_image);
}

void apply_blur_filter_gpu(animated_gif *image, int size, int threshold) {

}

void apply_sobel_filter_gpu(animated_gif *image) {
    /**
     * Almost same code than apply_gray_filter_gpu
     **/ 
     int print_time = 1;
     struct timeval t1, t2;
 
     int im_num;
     int width = image->width[0];
     int height = image->height[0];
     int size = width*height;
 
     /**
      * Allocation on the device once for all the images (if multiple)
      * Memory allocation + dimension of grid
      **/
     if (print_time) gettimeofday(&t1, NULL);
 
     pixel *device_image, *device_new;
     hipMalloc(&device_image, size*sizeof(pixel));
     hipMalloc(&device_new, size*sizeof(pixel));
 
     // Image cut into n*m rectangles
     int n = width / 1024 + 1;
     int m = height / 1024 + 1;
     dim3 dimGrid(n, m);
     dim3 dimBlock(width / n + 1, height / m + 1);
 
     if (print_time) {
         gettimeofday(&t2, NULL);
         printf("Alloc done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
         gettimeofday(&t1, NULL); 
     }
 
     /**
      * Memory transfert + computation
      **/
     for (im_num = 0; im_num < image->n_images; im_num++) {
         // Memory transfer
         hipMemcpy(device_image, image->p[im_num], size * sizeof(pixel), hipMemcpyHostToDevice);
         if (print_time) {
             gettimeofday(&t2, NULL);
             printf("Transfer done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
             gettimeofday(&t1, NULL); 
         }
 
         // Computation
         sobel<<<dimGrid, dimBlock>>>(device_image, device_new, height, width);
         if (print_time) {
             gettimeofday(&t2, NULL);
             printf("Computation done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
             gettimeofday(&t1, NULL); 
         }
 
         // Transfer back
         hipMemcpy(image->p[im_num], device_new, size * sizeof(pixel), hipMemcpyHostToDevice);
         if (print_time) {
             gettimeofday(&t2, NULL);
             printf("Transfer back done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
             gettimeofday(&t1, NULL); 
         }
 
     }
 
     hipFree(device_image);
     hipFree(device_new);
 }