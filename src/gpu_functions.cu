#include "hip/hip_runtime.h"
/**
 * File containing the 3 apply-filter functions implemented 
 * on the GPU using CUDA.
 */

#include <string.h>
#include <stdio.h>
#include <stdlib.h>

#include <datatype.h>

/********************************************************
 * Device functions *************************************
 ********************************************************/
__global__ void gray() {

}

__global__ void blur() {

}

__global__ void sobel() {
    
}

/********************************************************
 * Host functions****************************************
 ********************************************************/

void apply_gray_filter_gpu(animated_gif *image) {

}

void apply_blur_filter_gpu(animated_gif *image, int size, int threshold) {

}

void apply_sobel_filter_gpu(animated_gif *image) {

}