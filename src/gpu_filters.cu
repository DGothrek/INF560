#include "hip/hip_runtime.h"
/**
 * File containing the 3 apply-filter functions implemented 
 * on the GPU using CUDA.
 */

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <sys/time.h>

#include <filters.h>

/********************************************************
 * Device functions *************************************
 ********************************************************/
__global__ void gray(pixel *im, int height, int width)
{
  // int i, j;
  int moy, pos;
  // j = threadIdx.x + blockIdx.x * blockDim.x;
  // i = threadIdx.y + blockIdx.y * blockDim.y;
  pos = threadIdx.x + blockIdx.x * blockDim.x;
  // int id = threadIdx.x
  //         + blockDim.x*threadIdx.y
  //         + blockIdx.x*blockDim.x*blockDim.y
  //         + blockIdx.y*blockDim.x*blockDim.y*gridDim.x;

  // pos = i + j * width;
  // if (i < height && j < width)
  if (pos < width * height)
  {
    moy = (im[pos].r + im[pos].g + im[pos].b) / 3;
    if (moy < 0)
      moy = 0;
    if (moy > 255)
      moy = 255;

    im[pos].r = moy;
    im[pos].g = moy;
    im[pos].b = moy;
  }
}

__global__ void blur()
{
}

__global__ void sobel(pixel *im, pixel *im_new, int height, int width)
{
  int i, j, pos;

  pos = threadIdx.x + blockIdx.x * blockDim.x;
  i = pos / width;
  j = pos % width;

  int pixel_blue_no, pixel_blue_n, pixel_blue_ne;
  int pixel_blue_so, pixel_blue_s, pixel_blue_se;
  int pixel_blue_o, pixel_blue_e;

  float deltaX_blue;
  float deltaY_blue;
  float val_blue;

  if (i > 1 && i < height - 1 && j > 1 && j < width - 1)
  {
    pixel_blue_no = im[CONV(i - 1, j - 1, width)].b;
    pixel_blue_n = im[CONV(i - 1, j, width)].b;
    pixel_blue_ne = im[CONV(i - 1, j + 1, width)].b;
    pixel_blue_so = im[CONV(i + 1, j - 1, width)].b;
    pixel_blue_s = im[CONV(i + 1, j, width)].b;
    pixel_blue_se = im[CONV(i + 1, j + 1, width)].b;
    pixel_blue_o = im[CONV(i, j - 1, width)].b;
    pixel_blue_e = im[CONV(i, j + 1, width)].b;

    deltaX_blue = -pixel_blue_no + pixel_blue_ne - 2 * pixel_blue_o + 2 * pixel_blue_e - pixel_blue_so + pixel_blue_se;
    deltaY_blue = pixel_blue_se + 2 * pixel_blue_s + pixel_blue_so - pixel_blue_ne - 2 * pixel_blue_n - pixel_blue_no;
    val_blue = sqrt(deltaX_blue * deltaX_blue + deltaY_blue * deltaY_blue) / 4;

    if (val_blue > 50)
    {
        im_new[CONV(i, j, width)].r = 255;
        im_new[CONV(i, j, width)].g = 255;
        im_new[CONV(i, j, width)].b = 255;
    }
    else
    {
        im_new[CONV(i, j, width)].r = 0;
        im_new[CONV(i, j, width)].g = 0;
        im_new[CONV(i, j, width)].b = 0;
    }
  }

  else
  { 
    if (i < height && j < width) {
      im_new[CONV(i, j, width)] = im[CONV(i, j, width)];
    }
  }
}

/********************************************************
 * Host functions****************************************
 ********************************************************/
extern "C"
{
  void apply_gray_filter_gpu(animated_gif *image)
  {
    /**
     * Assuming images of same size in a multiple image GIF
     **/
    int print_time = 1;
    struct timeval t1, t2;

    int im_num;
    int width = image->width[0];
    int height = image->height[0];
    int size = width * height;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    /**
     * Allocation on the device once for all the images (if multiple)
     * Memory allocation + dimension of grid
     **/
    if (print_time)
      gettimeofday(&t1, NULL);

    pixel *device_image;
    hipMalloc(&device_image, size * sizeof(pixel));

    // Image cut into n*m rectangles
    // int n = width / deviceProp.maxThreadsPerBlock + 1;
    // int m = height / deviceProp.maxThreadsPerBlock + 1;
    // int n = width / 32 + 1;
    // int m = height / 32 + 1;
    // dim3 dimGrid(n, m);
    // dim3 dimBlock(width / n + 1, height / m + 1);
    dim3 dimGrid(size / deviceProp.maxThreadsPerBlock + 1);
    dim3 dimBlock(deviceProp.maxThreadsPerBlock);

    if (print_time)
    {
      gettimeofday(&t2, NULL);
      printf("Alloc done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
      gettimeofday(&t1, NULL);
    }

    /**
     * Memory transfert + computation
     **/
    for (im_num = 0; im_num < image->n_images; im_num++)
    {
      // Memory transfer
      hipMemcpy(device_image, image->p[im_num], size * sizeof(pixel), hipMemcpyHostToDevice);
      if (print_time)
      {
        gettimeofday(&t2, NULL);
        printf("Transfer done in %ld us with ", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
        printf(hipGetErrorString(hipGetLastError()));
        printf("\n");
        gettimeofday(&t1, NULL);
      }



      // Computation
      gray<<<dimGrid, dimBlock>>>(device_image, height, width);

      if (print_time)
      {
        gettimeofday(&t2, NULL);
        printf("Computation done in %ld us with ", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
        printf(hipGetErrorString(hipGetLastError()));
        printf("\n");
        gettimeofday(&t1, NULL);
      }

      // Transfer back
      hipMemcpy(image->p[im_num], device_image, size * sizeof(pixel), hipMemcpyDeviceToHost);
      if (print_time)
      {
        gettimeofday(&t2, NULL);
        printf("Transfer back done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
        gettimeofday(&t1, NULL);
      }
    }

    hipFree(device_image);
  }

  void apply_blur_filter_gpu(animated_gif *image, int size, int threshold)
  {
  }

  void apply_sobel_filter_gpu(animated_gif *image)
  {
    /**
     * Almost same code than apply_gray_filter_gpu
     **/
    int print_time = 1;
    struct timeval t1, t2;

    int im_num;
    int width = image->width[0];
    int height = image->height[0];
    int size = width * height;
    // printf("Size of the image = %d\n", size);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    /**
      * Allocation on the device once for all the images (if multiple)
      * Memory allocation + dimension of grid
      **/
    if (print_time)
      gettimeofday(&t1, NULL);

    pixel *device_image, *device_new;
    hipMalloc(&device_image, size * sizeof(pixel));
    hipMalloc(&device_new, size * sizeof(pixel));

    dim3 dimGrid(size / deviceProp.maxThreadsPerBlock + 1);
    dim3 dimBlock(deviceProp.maxThreadsPerBlock);


    if (print_time)
    {
      gettimeofday(&t2, NULL);
      printf("Alloc done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
      gettimeofday(&t1, NULL);
    }

    /**
      * Memory transfert + computation
      **/
    for (im_num = 0; im_num < image->n_images; im_num++)
    {
      // Memory transfer
      hipMemcpy(device_image, image->p[im_num], size * sizeof(pixel), hipMemcpyHostToDevice);
      if (print_time)
      {
        gettimeofday(&t2, NULL);
        printf("Transfer done in %ld us with ", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
        printf(hipGetErrorString(hipGetLastError()));
        printf("\n");
        gettimeofday(&t1, NULL);
      }

      // Computation
      sobel<<<dimGrid, dimBlock>>>(device_image, device_new, height, width);

      if (print_time)
      {
        gettimeofday(&t2, NULL);
        printf("Computation done in %ld us with ", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
        printf(hipGetErrorString(hipGetLastError()));
        printf("\n");
        gettimeofday(&t1, NULL);
      }

      // Transfer back
      hipMemcpy(image->p[im_num], device_new, size * sizeof(pixel), hipMemcpyDeviceToHost);
      if (print_time)
      {
        gettimeofday(&t2, NULL);
        printf("Transfer back done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
        gettimeofday(&t1, NULL);
      }
    }

    hipFree(device_image);
    hipFree(device_new);
  }
}