
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

#include <sys/time.h>

__global__ void vectAdd(int *a, int *b, int *c, int len)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < len)
  {
    c[i] = a[i] + b[i];
  }
}

/* Function computing the final string to print */
void vector_add(int *c, int *a, int *b, int length)
{
  int *d_a, *d_b, *d_c;
  struct timeval t1, t2;

  hipMalloc(&d_a, length * sizeof(int));
  hipMalloc(&d_b, length * sizeof(int));
  hipMalloc(&d_c, length * sizeof(int));

  gettimeofday(&t1, NULL);

  hipMemcpy(d_a, a, length * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, length * sizeof(int), hipMemcpyHostToDevice);

  gettimeofday(&t2, NULL);
  printf("Transfer done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
  gettimeofday(&t1, NULL);

  // a (1D/2D/3D) grid containing blocks, each one containing threads.
  dim3 dimGrid(length / 1024 + 1);
  dim3 dimBlock(1024);
  vectAdd<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, length);

  gettimeofday(&t2, NULL);
  printf("Processing done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
  gettimeofday(&t1, NULL);

  hipMemcpy(c, d_c, length * sizeof(int), hipMemcpyDeviceToHost);

  gettimeofday(&t2, NULL);
  printf("Tranfer back done in %ld us\n", (t2.tv_sec - t1.tv_sec) * 1000000 + (t2.tv_usec - t1.tv_usec));
  gettimeofday(&t1, NULL);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

int main(int argc, char **argv)
{
  int N, S;

  if (argc < 2)
  {
    printf("Usage: %s N S\n", argv[0]);
    printf("\tS: seed for pseudo-random generator\n");
    printf("\tN: size of the array\n");
    exit(1);
  }

  N = atoi(argv[1]);
  S = atoi(argv[2]);
  srand48(S);

  int *A, *B, *C;
  int i;
  A = (int *)malloc(N * sizeof(int));
  B = (int *)malloc(N * sizeof(int));
  C = (int *)malloc(N * sizeof(int));

  /* Initialize the array */
  for (i = 0; i < N; i++)
  {
    A[i] = lrand48();
    B[i] = lrand48();
  }

  vector_add(C, A, B, N);

  /* Checking the result */
  printf("Checking the result...\n");
  for (i = 0; i < N; i++)
  {
    if (C[i] != A[i] + B[i])
    {
      printf("Wrong res for i=%d\n", i);
      return 0;
    }
  }

  printf("Res OK!\n");

  return 0;
}
