
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void vectAdd(char *a, char *b, char *c, char *res, int len)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;

  res[i] = a[i] + b[i] + c[i];
}

/* Function computing the final string to print */
void compute_string(char *res, char *a, char *b, char *c, int length)
{
  char *d_a, *d_b, *d_c, *d_res;

  hipMalloc(&d_a, length * sizeof(char));
  hipMalloc(&d_b, length * sizeof(char));
  hipMalloc(&d_c, length * sizeof(char));
  hipMalloc(&d_res, length * sizeof(char));

  hipMemcpy(d_a, a, length * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, length * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, length * sizeof(char), hipMemcpyHostToDevice);

  dim3 dimBlock(30);
  dim3 dimGrid(1);
  vectAdd<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, d_res, length);

  hipMemcpy(res, d_res, length * sizeof(char), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_res);
}


int main()
{

  char *res;

  char a[30] = {40, 70, 70, 70, 80, 0, 50, 80, 80, 70, 70, 0, 40, 80, 79,
                70, 0, 40, 50, 50, 0, 70, 80, 0, 30, 50, 30, 30, 0, 0};
  char b[30] = {10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10,
                10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 0, 0};
  char c[30] = {22, 21, 28, 28, 21, 22, 27, 21, 24, 28, 20, 22, 20, 24, 22,
                29, 22, 21, 20, 25, 22, 25, 20, 22, 27, 25, 28, 25, 0, 0};

  res = (char *)malloc(30 * sizeof(char));

  /* This function call should be programmed in CUDA */
  /* -> need to allocate and transfer data to/from the device */
  compute_string(res, a, b, c, 30);

  printf("%s\n", res);

  return 0;
}
